#include "hip/hip_runtime.h"
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include <hip/hip_runtime_api.h>
#include <cuda/Orb.hpp>
#include <Utils.hpp>

using namespace cv;
using namespace cv::cuda;
using namespace cv::cuda::device;

namespace Orb {

#define GET_VALUE(idx) \
    image(loc.y + __float2int_rn(pattern[idx].x * b + pattern[idx].y * a), \
          loc.x + __float2int_rn(pattern[idx].x * a - pattern[idx].y * b))

  __global__ void calcOrb_kernel(const PtrStepb image, KeyPoint * keypoints, const int npoints, const Point * pattern, PtrStepb descriptors) {
    int id = blockIdx.x;
    int tid = threadIdx.x;
    if (id >= npoints) return;

    const KeyPoint &kpt = keypoints[id];
    short2 loc = make_short2(kpt.pt.x, kpt.pt.y);
    pattern += 16 * tid;

    uchar * desc = descriptors.ptr(id);
    const float factorPI = (float)(CV_PI/180.f);
    float angle = (float)kpt.angle * factorPI;
    float a = (float)cosf(angle), b = (float)sinf(angle);

    int t0, t1, val;
    t0 = GET_VALUE(0); t1 = GET_VALUE(1);
    val = t0 < t1;
    t0 = GET_VALUE(2); t1 = GET_VALUE(3);
    val |= (t0 < t1) << 1;
    t0 = GET_VALUE(4); t1 = GET_VALUE(5);
    val |= (t0 < t1) << 2;
    t0 = GET_VALUE(6); t1 = GET_VALUE(7);
    val |= (t0 < t1) << 3;
    t0 = GET_VALUE(8); t1 = GET_VALUE(9);
    val |= (t0 < t1) << 4;
    t0 = GET_VALUE(10); t1 = GET_VALUE(11);
    val |= (t0 < t1) << 5;
    t0 = GET_VALUE(12); t1 = GET_VALUE(13);
    val |= (t0 < t1) << 6;
    t0 = GET_VALUE(14); t1 = GET_VALUE(15);
    val |= (t0 < t1) << 7;

    desc[tid] = (uchar)val;
  }

  //void computeOrbDescriptors(const Mat& image, vector<KeyPoint>& keypoints, Mat& descriptors, const vector<Point>& pattern)
  void computeOrbDescriptors(InputArray _image, const KeyPoint * _keypoints, const int npoints, Mat & _descriptors, const Point * _pattern) {
    PUSH_RANGE("computeDescriptors", 1);
    if (npoints == 0) {
      POP_RANGE;
      return ;
    }
    hipStream_t stream = 0;
    const GpuMat image = _image.getGpuMat();
    KeyPoint * keypoints;
    checkCudaErrors( hipMalloc(&keypoints, sizeof(KeyPoint) * npoints) );
    checkCudaErrors( hipMemcpyAsync(keypoints, _keypoints, sizeof(KeyPoint) * npoints, hipMemcpyHostToDevice, stream) );
    Point * pattern;
    checkCudaErrors( hipMalloc(&pattern, sizeof(Point) * 512) );
    checkCudaErrors( hipMemcpyAsync(pattern, _pattern, sizeof(Point) * 512, hipMemcpyHostToDevice, stream) );

    GpuMat descriptors(npoints, 32, CV_8UC1, Scalar::all(0));

    dim3 dimBlock(32);
    dim3 dimGrid(npoints);
    calcOrb_kernel<<<dimGrid, dimBlock, 0, stream>>>(image, keypoints, npoints, pattern, descriptors);
    checkCudaErrors( hipGetLastError() );

    checkCudaErrors( hipStreamSynchronize(stream) );

    descriptors.download(_descriptors);
    checkCudaErrors( hipFree(keypoints) );
    POP_RANGE;
  }

}
